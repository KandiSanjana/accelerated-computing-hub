#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void kernel(int *data1, int *data2) {
  data1[threadIdx.x] = threadIdx.x;
  data2[threadIdx.x] = threadIdx.x;
}

int main() {
  int device_id;
  hipGetDevice(&device_id);

  int *data = (int *)malloc(128 * sizeof(int));
  hipMemLocation loc;
  loc.type = hipMemLocationTypeHost;
  cudaMemAdvise_v2(data, 128 * sizeof(int), hipMemAdviseSetPreferredLocation,
                   loc);
  cudaMemPrefetchAsync_v2(data, 128 * sizeof(int), loc, 0);

  int data2[128];
  hipMemLocation loc2;
  loc2.id = device_id;
  loc2.type = hipMemLocationTypeDevice;
  cudaMemAdvise_v2(data2, 128 * sizeof(int), hipMemAdviseSetPreferredLocation,
                   loc2);
  cudaMemPrefetchAsync_v2(data2, 128 * sizeof(int), loc2, 0);

  kernel<<<1, 128>>>(data, data2);
  hipDeviceSynchronize();

  for (int i = 0; i < 128; i++) {
    printf("(%d, %d)  ", data[i], data2[i]);
  }
  printf("\n");

  free(data);
  return 0;
}

#include <cstdlib>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *data)
{
    data[threadIdx.x] += 2;
}
int main()
{
    int N = 128;
    int *data = (int *)malloc(N * sizeof(int));
    int *d_data;

    hipMalloc((void **)&d_data, N * sizeof(int));

    hipMemcpy(d_data, data, N * sizeof(int), hipMemcpyHostToDevice);

    kernel<<<1, N>>>(d_data);

    hipMemcpy(d_data, data, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(d_data);
    free(data);
}

